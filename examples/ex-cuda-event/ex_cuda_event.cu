#include "hip/hip_runtime.h"
// MIT License
//
// Copyright (c) 2019, The Regents of the University of California,
// through Lawrence Berkeley National Laboratory (subject to receipt of any
// required approvals from the U.S. Dept. of Energy).  All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//

#include <cassert>
#include <chrono>
#include <cmath>
#include <fstream>
#include <future>
#include <iomanip>
#include <iterator>
#include <random>
#include <thread>
#include <unordered_map>
#include <vector>

#include "timemory/ert/kernels.hpp"
#include "timemory/timemory.hpp"
#include "timemory/utility/signals.hpp"
#include "timemory/utility/testing.hpp"

#if defined(TIMEMORY_USE_CUPTI)
#    include "timemory/components/cupti/cupti_counters.hpp"
#endif

using namespace tim::component;

// using papi_tuple_t = papi_tuple<PAPI_TOT_CYC, PAPI_TOT_INS, PAPI_BR_MSP, PAPI_BR_PRC>;
using auto_tuple_t = tim::auto_tuple_t<wall_clock, system_clock, cpu_clock, cpu_util,
                                       nvtx_marker, papi_array_t>;
using comp_tuple_t = typename auto_tuple_t::component_type;
using cuda_tuple_t = tim::auto_tuple_t<cuda_event, nvtx_marker>;
using counter_t    = ert_timer;
using ert_data_t   = tim::ert::exec_data<counter_t>;

//======================================================================================//

#define CUDA_CHECK_LAST_ERROR()                                                          \
    {                                                                                    \
        tim::cuda::stream_sync(0);                                                       \
        hipError_t err = hipGetLastError();                                              \
        if(hipSuccess != err)                                                           \
        {                                                                                \
            fprintf(stderr, "cudaCheckError() failed at %s@'%s':%i : %s\n",              \
                    __FUNCTION__, __FILE__, __LINE__, hipGetErrorString(err));          \
            std::stringstream ss;                                                        \
            ss << "cudaCheckError() failed at " << __FUNCTION__ << "@'" << __FILE__      \
               << "':" << __LINE__ << " : " << hipGetErrorString(err);                  \
            throw std::runtime_error(ss.str());                                          \
        }                                                                                \
    }

//======================================================================================//

template <typename Tp>
std::string
array_to_string(const Tp& arr, const std::string& delimiter = ", ",
                const int& _width = 16, const int& _break = 8,
                const std::string& _break_delim = "\t")
{
    auto size      = std::distance(arr.begin(), arr.end());
    using int_type = decltype(size);
    std::stringstream ss;
    for(int_type i = 0; i < size; ++i)
    {
        ss << std::setw(_width) << arr.at(i);
        if(i + 1 < size)
            ss << delimiter;
        if((i + 1) % _break == 0 && (i + 1) < size)
            ss << "\n" << _break_delim;
    }
    return ss.str();
}

//--------------------------------------------------------------------------------------//

static const int nitr = 10;
static int64_t   N    = 50 * (1 << 23);
static auto      Nsub = N / nitr;

//--------------------------------------------------------------------------------------//
// saxpy calculation
TIMEMORY_GLOBAL_FUNCTION void
warmup(int64_t n)
{
    int i   = blockIdx.x * blockDim.x + threadIdx.x;
    int tmp = 0;
    if(i < n)
        tmp += i;
}

//--------------------------------------------------------------------------------------//
// saxpy calculation
TIMEMORY_GLOBAL_FUNCTION void
saxpy(int64_t n, float a, float* x, float* y)
{
    auto itr = tim::device::grid_strided_range<tim::device::default_device, 0>(n);
    for(int i = itr.begin(); i < itr.end(); i += itr.stride())
        y[i] = a * x[i] + y[i];
}
//--------------------------------------------------------------------------------------//

void
warmup()
{
    int     block = 128;
    int     ngrid = 128;
    int64_t val   = 256;
    warmup<<<ngrid, block>>>(val);
    // CUDA_CHECK_LAST_ERROR();
}

//======================================================================================//

void
print_info(const std::string&);
void
print_string(const std::string& str);
void
test_1_saxpy();
void
test_2_saxpy_async();
void
test_3_saxpy_pinned();
void
test_4_saxpy_async_pinned();
void
test_5_mt_saxpy_async();
void
test_6_mt_saxpy_async_pinned();
void
test_7_cupti_available();
void
test_8_cupti_subset();
void
test_9_cupti_counters();
void
test_10_cupti_metric();

//======================================================================================//

int
main(int argc, char** argv)
{
    if(N % nitr != 0)
    {
        throw std::runtime_error("Error N is not a multiple of nitr");
    }

    tim::settings::timing_scientific() = true;
    tim::timemory_init(argc, argv);
    tim::settings::json_output() = true;
    tim::enable_signal_detection();

    int ndevices = tim::cuda::device_count();
    warmup();

    auto* timing = new tim::component_tuple<wall_clock, system_clock, cpu_clock, cpu_util,
                                            nvtx_marker>("Tests runtime", true);

    timing->start();

    CONFIGURE_TEST_SELECTOR(10);

    int num_fail = 0;
    int num_test = 0;

    if(ndevices == 0)
    {
        for(auto i : { 3, 4, 6 })
        {
            if(tests.count(i) > 0)
                tests.erase(tests.find(i));
        }
    }

    std::cout << "# tests: " << tests.size() << std::endl;
    try
    {
        RUN_TEST(1, test_1_saxpy, num_test, num_fail);
        RUN_TEST(2, test_2_saxpy_async, num_test, num_fail);
        RUN_TEST(3, test_3_saxpy_pinned, num_test, num_fail);
        RUN_TEST(4, test_4_saxpy_async_pinned, num_test, num_fail);
        RUN_TEST(5, test_5_mt_saxpy_async, num_test, num_fail);
        RUN_TEST(6, test_6_mt_saxpy_async_pinned, num_test, num_fail);
        RUN_TEST(7, test_7_cupti_available, num_test, num_fail);
        RUN_TEST(8, test_8_cupti_subset, num_test, num_fail);
        RUN_TEST(9, test_9_cupti_counters, num_test, num_fail);
        RUN_TEST(10, test_10_cupti_metric, num_test, num_fail);
    } catch(std::exception& e)
    {
        std::cerr << e.what() << std::endl;
    }

    timing->stop();
    std::cout << "\n" << *timing << std::endl;

    TEST_SUMMARY(argv[0], num_test, num_fail);
    delete timing;

    tim::timemory_finalize();
    exit(num_fail);
}

//======================================================================================//

void
print_info(const std::string& func)
{
    if(tim::dmp::rank() == 0)
    {
        std::cout << "\n[" << tim::dmp::rank() << "]\e[1;33m TESTING \e[0m["
                  << "\e[1;36m" << func << "\e[0m"
                  << "]...\n"
                  << std::endl;
    }
}

//======================================================================================//

void
print_string(const std::string& str)
{
    std::stringstream _ss;
    _ss << "[" << tim::dmp::rank() << "] " << str << std::endl;
    std::cout << _ss.str();
}

//======================================================================================//

void
test_1_saxpy()
{
    print_info(__FUNCTION__);
    warmup();
    TIMEMORY_BASIC_MARKER(auto_tuple_t, "");

    comp_tuple_t _clock("Runtime");
    _clock.start();

    float*      x;
    float*      y;
    float*      d_x;
    float*      d_y;
    int         block    = 512;
    int         ngrid    = (N + block - 1) / block;
    float       nseconds = 0.0f;
    float       maxError = 0.0f;
    float       sumError = 0.0f;
    cuda_event* evt      = nullptr;

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[cpu_malloc]");
        x = tim::device::cpu::alloc<float>(N);
        y = tim::device::cpu::alloc<float>(N);
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[gpu_malloc]");
        d_x = tim::cuda::malloc<float>(N);
        d_y = tim::cuda::malloc<float>(N);
        CUDA_CHECK_LAST_ERROR();
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[assign]");
        for(int i = 0; i < N; i++)
        {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[create_event]");
        evt = new cuda_event();
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[H2D]");
        evt->mark_begin();
        tim::cuda::memcpy(d_x, x, N, tim::cuda::host_to_device_v);
        tim::cuda::memcpy(d_y, y, N, tim::cuda::host_to_device_v);
        evt->mark_end();
        CUDA_CHECK_LAST_ERROR();
    }

    for(int i = 0; i < 1; ++i)
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[", i, "]");
        // Perform SAXPY on 1M elements
        evt->mark_begin();
        saxpy<<<ngrid, block>>>(N, 1.0f, d_x, d_y);
        evt->mark_end();
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[D2H]");
        evt->mark_begin();
        hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
        evt->mark_end();
    }

    tim::cuda::device_sync();

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[check]");
        for(int64_t i = 0; i < N; i++)
        {
            maxError = std::max(maxError, std::abs(y[i] - 2.0f));
            sumError += std::abs(y[i] - 2.0f);
        }
    }

    evt->sync();
    nseconds += evt->get();
    _clock.stop();
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[output]");
        std::cout << "Event: " << *evt << std::endl;
        std::cout << _clock << std::endl;
        printf("Max error: %8.4e\n", maxError);
        printf("Sum error: %8.4e\n", sumError);
        printf("Effective Bandwidth (GB/s): %f\n",
               N * 4 * 3 / nseconds / tim::units::gigabyte);
        printf("Kernel Runtime (sec): %16.12e\n", nseconds);
    }

    delete evt;

    tim::device::gpu::free(d_x);
    tim::device::gpu::free(d_y);
    tim::device::cpu::free(x);
    tim::device::cpu::free(y);

    tim::cuda::device_sync();
    // tim::cuda::device_reset();
}

//======================================================================================//

void
test_2_saxpy_async()
{
    print_info(__FUNCTION__);
    warmup();
    TIMEMORY_BASIC_MARKER(auto_tuple_t, "");

    comp_tuple_t _clock("Runtime");
    _clock.start();

    float*        x;
    float*        y;
    float*        d_x;
    float*        d_y;
    int           block    = 512;
    int           ngrid    = (Nsub + block - 1) / block;
    float         nseconds = 0.0f;
    float         maxError = 0.0f;
    float         sumError = 0.0f;
    cuda_event**  evt      = new cuda_event*[nitr];
    hipStream_t* stream   = new hipStream_t[nitr];

    auto _sync = [&]() {
        for(int i = 0; i < nitr; i++)
            tim::cuda::stream_sync(stream[i]);
    };

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[cpu_malloc]");
        x = tim::device::cpu::alloc<float>(N);
        y = tim::device::cpu::alloc<float>(N);
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[gpu_malloc]");
        d_x = tim::cuda::malloc<float>(N);
        d_y = tim::cuda::malloc<float>(N);
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[assign]");
        for(int i = 0; i < N; i++)
        {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[create]");
        for(int i = 0; i < nitr; ++i)
        {
            tim::cuda::stream_create(stream[i]);
            evt[i] = new cuda_event(stream[i]);
            evt[i]->start();
        }
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[H2D]");
        for(int i = 0; i < nitr; ++i)
        {
            auto   offset = Nsub * i;
            float* _x     = x + offset;
            float* _y     = y + offset;
            float* _dx    = d_x + offset;
            float* _dy    = d_y + offset;
            evt[i]->mark_begin();
            tim::cuda::memcpy(_dx, _x, Nsub, tim::cuda::host_to_device_v, stream[i]);
            tim::cuda::memcpy(_dy, _y, Nsub, tim::cuda::host_to_device_v, stream[i]);
            evt[i]->mark_end();
        }
    }

    for(int i = 0; i < nitr; ++i)
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[", i, "]");

        auto   offset = Nsub * i;
        float* _dx    = d_x + offset;
        float* _dy    = d_y + offset;

        evt[i]->mark_begin();
        // Perform SAXPY on 1M elements
        saxpy<<<ngrid, block, 0, stream[i]>>>(Nsub, 1.0f, _dx, _dy);
        evt[i]->mark_end();
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[D2H]");
        for(int i = 0; i < nitr; ++i)
        {
            auto   offset = Nsub * i;
            float* _y     = y + offset;
            float* _dy    = d_y + offset;
            evt[i]->mark_begin();
            tim::cuda::memcpy(_y, _dy, Nsub, tim::cuda::device_to_host_v, stream[i]);
            evt[i]->mark_end();
        }
    }

    _sync();

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[check]");
        for(int64_t i = 0; i < N; i++)
        {
            maxError = std::max(maxError, std::abs(y[i] - 2.0f));
            sumError += std::abs(y[i] - 2.0f);
        }
    }

    _clock.stop();
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[output]");
        cuda_event _evt = **evt;
        for(int i = 1; i < nitr; ++i)
        {
            evt[i]->stop();
            nseconds += evt[i]->get();
            _evt += *(evt[i]);
        }
        std::cout << "Event: " << _evt << std::endl;
        std::cout << _clock << std::endl;
        printf("Max error: %8.4e\n", maxError);
        printf("Sum error: %8.4e\n", sumError);
        printf("Effective Bandwidth (GB/s): %f\n",
               N * 4 * 3 / nseconds / tim::units::gigabyte);
        printf("Kernel Runtime (sec): %16.12e\n", nseconds);
    }

    for(int i = 0; i < nitr; ++i)
        delete evt[i];
    delete[] evt;

    tim::device::gpu::free(d_x);
    tim::device::gpu::free(d_y);
    tim::device::cpu::free(x);
    tim::device::cpu::free(y);

    tim::cuda::device_sync();
}

//======================================================================================//

void
test_3_saxpy_pinned()
{
    print_info(__FUNCTION__);
    warmup();
    TIMEMORY_BASIC_MARKER(auto_tuple_t, "");

    comp_tuple_t _clock("Runtime");
    _clock.start();

    float*      x;
    float*      y;
    float*      d_x;
    float*      d_y;
    int         block    = 512;
    int         ngrid    = (N + block - 1) / block;
    float       nseconds = 0.0f;
    float       maxError = 0.0f;
    float       sumError = 0.0f;
    cuda_event* evt      = nullptr;

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[cpu_malloc]");
        x = tim::cuda::malloc_host<float>(N);
        y = tim::cuda::malloc_host<float>(N);
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[gpu_malloc]");
        d_x = tim::cuda::malloc<float>(N);
        d_y = tim::cuda::malloc<float>(N);
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[assign]");
        for(int i = 0; i < N; i++)
        {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[create_event]");
        evt = new cuda_event();
        evt->start();
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[H2D]");
        evt->mark_begin();
        tim::cuda::memcpy(d_x, x, N, tim::cuda::host_to_device_v);
        tim::cuda::memcpy(d_y, y, N, tim::cuda::host_to_device_v);
        evt->mark_end();
    }

    for(int i = 0; i < 1; ++i)
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[", i, "]");
        evt->mark_begin();
        // Perform SAXPY on 1M elements
        saxpy<<<ngrid, block>>>(N, 1.0f, d_x, d_y);
        evt->mark_end();
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[D2H]");
        evt->mark_begin();
        tim::cuda::memcpy(y, d_y, N, tim::cuda::device_to_host_v);
        evt->mark_end();
    }

    tim::cuda::device_sync();

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[check]");
        for(int64_t i = 0; i < N; i++)
        {
            maxError = std::max(maxError, std::abs(y[i] - 2.0f));
            sumError += std::abs(y[i] - 2.0f);
        }
    }

    _clock.stop();
    evt->stop();
    nseconds += evt->get();
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[output]");
        std::cout << "Event: " << *evt << std::endl;
        std::cout << _clock << std::endl;
        printf("Max error: %8.4e\n", maxError);
        printf("Sum error: %8.4e\n", sumError);
        printf("Effective Bandwidth (GB/s): %f\n",
               N * 4 * 3 / nseconds / tim::units::gigabyte);
        printf("Kernel Runtime (sec): %16.12e\n", nseconds);
    }

    delete evt;

    tim::cuda::free_host(x);
    tim::cuda::free_host(y);
    tim::cuda::free(d_x);
    tim::cuda::free(d_y);

    tim::cuda::device_sync();
    // tim::cuda::device_reset();
}

//======================================================================================//

void
test_4_saxpy_async_pinned()
{
    print_info(__FUNCTION__);
    warmup();
    TIMEMORY_BASIC_MARKER(auto_tuple_t, "");

    comp_tuple_t _clock("Runtime");
    _clock.start();

    float*        x;
    float*        y;
    float*        d_x;
    float*        d_y;
    int           block    = 512;
    int           ngrid    = (Nsub + block - 1) / block;
    float         nseconds = 0.0f;
    float         maxError = 0.0f;
    float         sumError = 0.0f;
    cuda_event**  evt      = new cuda_event*[nitr];
    hipStream_t* stream   = new hipStream_t[nitr];

    auto _sync = [&]() {
        for(int i = 0; i < nitr; i++)
            tim::cuda::stream_sync(stream[i]);
    };

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[cpu_malloc]");
        x = tim::cuda::malloc_host<float>(N);
        y = tim::cuda::malloc_host<float>(N);
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[gpu_malloc]");
        d_x = tim::cuda::malloc<float>(N);
        d_y = tim::cuda::malloc<float>(N);
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[assign]");
        for(int i = 0; i < N; i++)
        {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[create]");
        for(int i = 0; i < nitr; ++i)
        {
            tim::cuda::stream_create(stream[i]);
            evt[i] = new cuda_event(stream[i]);
            evt[i]->start();
        }
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[H2D]");
        for(int i = 0; i < nitr; ++i)
        {
            auto   offset = Nsub * i;
            float* _x     = x + offset;
            float* _y     = y + offset;
            float* _dx    = d_x + offset;
            float* _dy    = d_y + offset;
            evt[i]->mark_begin();
            tim::cuda::memcpy(_dx, _x, Nsub, tim::cuda::host_to_device_v, stream[i]);
            tim::cuda::memcpy(_dy, _y, Nsub, tim::cuda::host_to_device_v, stream[i]);
            evt[i]->mark_end();
        }
    }

    for(int i = 0; i < nitr; ++i)
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[", i, "]");

        auto   offset = Nsub * i;
        float* _dx    = d_x + offset;
        float* _dy    = d_y + offset;

        evt[i]->mark_begin();
        // Perform SAXPY on 1M elements
        saxpy<<<ngrid, block, 0, stream[i]>>>(Nsub, 1.0f, _dx, _dy);
        evt[i]->mark_end();
    }

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[D2H]");
        for(int i = 0; i < nitr; ++i)
        {
            auto   offset = Nsub * i;
            float* _y     = y + offset;
            float* _dy    = d_y + offset;
            evt[i]->mark_begin();
            tim::cuda::memcpy(_y, _dy, Nsub, tim::cuda::device_to_host_v, stream[i]);
            evt[i]->mark_end();
        }
    }

    _sync();

    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[check]");
        for(int64_t i = 0; i < N; i++)
        {
            maxError = std::max(maxError, std::abs(y[i] - 2.0f));
            sumError += std::abs(y[i] - 2.0f);
        }
    }

    _clock.stop();
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[output]");
        cuda_event _evt = **evt;
        for(int i = 1; i < nitr; ++i)
        {
            evt[i]->stop();
            nseconds += evt[i]->get();
            _evt += *(evt[i]);
        }
        std::cout << "Event: " << _evt << std::endl;
        std::cout << _clock << std::endl;
        printf("Max error: %8.4e\n", maxError);
        printf("Sum error: %8.4e\n", sumError);
        printf("Effective Bandwidth (GB/s): %f\n",
               N * 4 * 3 / nseconds / tim::units::gigabyte);
        printf("Kernel Runtime (sec): %16.12e\n", nseconds);
    }

    for(int i = 0; i < nitr; ++i)
    {
        delete evt[i];
        tim::cuda::stream_destroy(stream[i]);
    }
    delete[] evt;

    tim::cuda::free(d_x);
    tim::cuda::free(d_y);
    tim::cuda::free_host(x);
    tim::cuda::free_host(y);

    tim::cuda::device_sync();
    // tim::cuda::device_reset();
}

//======================================================================================//

void
test_5_mt_saxpy_async()
{
    print_info(__FUNCTION__);
    warmup();
    TIMEMORY_BASIC_MARKER(auto_tuple_t, "");
    auto lambda_op = tim::string::join("", "::", _TIM_FUNC);

    comp_tuple_t _clock("Runtime");
    _clock.start();

    using data_t        = std::tuple<cuda_event, float, float, float>;
    using data_vector_t = std::vector<data_t>;

    data_vector_t data_vector(nitr);

    auto run_thread = [&](int i) {
        float*              x;
        float*              y;
        float*              d_x;
        float*              d_y;
        int                 block    = 512;
        int                 ngrid    = (Nsub + block - 1) / block;
        float               nseconds = 0.0f;
        float               maxError = 0.0f;
        float               sumError = 0.0f;
        tim::cuda::stream_t stream;
        tim::cuda::stream_create(stream);
        cuda_event evt(stream);
        evt.start();

        TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[run_thread]");

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[cpu_malloc]");
            x = tim::device::cpu::alloc<float>(Nsub);
            y = tim::device::cpu::alloc<float>(Nsub);
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[gpu_malloc]");
            d_x = tim::cuda::malloc<float>(Nsub);
            d_y = tim::cuda::malloc<float>(Nsub);
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[assign]");
            for(int i = 0; i < Nsub; i++)
            {
                x[i] = 1.0f;
                y[i] = 2.0f;
            }
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[H2D]");
            evt.mark_begin();
            tim::cuda::memcpy(d_x, x, Nsub, tim::cuda::host_to_device_v, stream);
            tim::cuda::memcpy(d_y, y, Nsub, tim::cuda::host_to_device_v, stream);
            evt.mark_end();
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[", i, "]");
            // Perform SAXPY on 1M elements
            evt.mark_begin();
            saxpy<<<ngrid, block, 0, stream>>>(Nsub, 1.0f, d_x, d_y);
            evt.mark_end();
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[D2H]");
            evt.mark_begin();
            tim::cuda::memcpy(y, d_y, Nsub, tim::cuda::device_to_host_v);
            evt.mark_end();
        }

        tim::cuda::stream_sync(stream);
        tim::cuda::stream_destroy(stream);

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[check]");
            for(int64_t i = 0; i < Nsub; i++)
            {
                maxError = std::max(maxError, std::abs(y[i] - 2.0f));
                sumError += std::abs(y[i] - 2.0f);
            }
        }

        evt.stop();
        nseconds += evt.get();
        data_vector[i] = std::make_tuple(evt, nseconds, maxError, sumError);

        tim::device::gpu::free(d_x);
        tim::device::gpu::free(d_y);
        tim::device::cpu::free(x);
        tim::device::cpu::free(y);
    };

    std::vector<std::thread> threads;
    for(int i = 0; i < nitr; i++)
        threads.emplace_back(run_thread, i);

    for(int i = 0; i < nitr; i++)
        threads[i].join();

    cuda_event evt      = std::get<0>(data_vector[0]);
    float      nseconds = std::get<1>(data_vector[0]);
    float      maxError = std::get<2>(data_vector[0]);
    float      sumError = std::get<3>(data_vector[0]);

    for(int i = 1; i < nitr; i++)
    {
        evt += std::get<0>(data_vector[i]);
        nseconds += std::get<1>(data_vector[i]);
        maxError = std::max(maxError, std::get<2>(data_vector[i]));
        sumError += std::get<3>(data_vector[i]);
    }

    _clock.stop();
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[output]");
        std::cout << "Event: " << evt << std::endl;
        std::cout << _clock << std::endl;
        printf("Max error: %8.4e\n", maxError);
        printf("Sum error: %8.4e\n", sumError);
        printf("Effective Bandwidth (GB/s): %f\n",
               N * 4 * 3 / nseconds / tim::units::gigabyte);
        printf("Kernel Runtime (sec): %16.12e\n", nseconds);
    }

    tim::cuda::device_sync();
    // tim::cuda::device_reset();
}

//======================================================================================//

void
test_6_mt_saxpy_async_pinned()
{
    print_info(__FUNCTION__);
    warmup();
    TIMEMORY_BASIC_MARKER(auto_tuple_t, "");
    auto lambda_op = tim::string::join("", "::", _TIM_FUNC);

    comp_tuple_t _clock("Runtime");
    _clock.start();

    using data_t        = std::tuple<cuda_event, float, float, float>;
    using data_vector_t = std::vector<data_t>;

    data_vector_t data_vector(nitr);

    auto run_thread = [&](int i) {
        float*              x;
        float*              y;
        float*              d_x;
        float*              d_y;
        int                 block    = 512;
        int                 ngrid    = (Nsub + block - 1) / block;
        float               nseconds = 0.0f;
        float               maxError = 0.0f;
        float               sumError = 0.0f;
        tim::cuda::stream_t stream;
        tim::cuda::stream_create(stream);
        cuda_event* evt = new cuda_event(stream);
        evt->start();

        TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[run_thread]");

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[cpu_malloc]");
            x = tim::cuda::malloc_host<float>(Nsub);
            y = tim::cuda::malloc_host<float>(Nsub);
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[gpu_malloc]");
            d_x = tim::cuda::malloc_host<float>(Nsub);
            d_y = tim::cuda::malloc_host<float>(Nsub);
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[assign]");
            for(int i = 0; i < Nsub; i++)
            {
                x[i] = 1.0f;
                y[i] = 2.0f;
            }
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[H2D]");
            evt->mark_begin();
            tim::cuda::memcpy(d_x, x, Nsub, tim::cuda::host_to_device_v, stream);
            tim::cuda::memcpy(d_y, y, Nsub, tim::cuda::host_to_device_v, stream);
            evt->mark_end();
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[", i, "]");

            // Perform SAXPY on 1M elements
            evt->mark_begin();
            saxpy<<<ngrid, block, 0, stream>>>(Nsub, 1.0f, d_x, d_y);
            evt->mark_end();
        }

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[D2H]");
            evt->mark_begin();
            tim::cuda::memcpy(y, d_y, Nsub, tim::cuda::device_to_host_v, stream);
            evt->mark_end();
        }

        tim::cuda::stream_sync(stream);
        tim::cuda::stream_destroy(stream);

        {
            TIMEMORY_BASIC_MARKER(auto_tuple_t, lambda_op, "[check]");
            for(int64_t i = 0; i < Nsub; i++)
            {
                maxError = std::max(maxError, std::abs(y[i] - 2.0f));
                sumError += std::abs(y[i] - 2.0f);
            }
        }

        evt->stop();
        nseconds += evt->get();

        data_vector[i] = std::make_tuple(*evt, nseconds, maxError, sumError);

        tim::cuda::free_host(x);
        tim::cuda::free_host(y);
        tim::cuda::free(d_x);
        tim::cuda::free(d_y);
        delete evt;
    };

    std::vector<std::thread> threads;
    for(int i = 0; i < nitr; i++)
        threads.emplace_back(run_thread, i);

    for(int i = 0; i < nitr; i++)
        threads[i].join();

    cuda_event evt      = std::get<0>(data_vector[0]);
    float      nseconds = std::get<1>(data_vector[0]);
    float      maxError = std::get<2>(data_vector[0]);
    float      sumError = std::get<3>(data_vector[0]);

    for(int i = 1; i < nitr; i++)
    {
        evt += std::get<0>(data_vector[i]);
        nseconds += std::get<1>(data_vector[i]);
        maxError = std::max(maxError, std::get<2>(data_vector[i]));
        sumError += std::get<3>(data_vector[i]);
    }

    _clock.stop();
    {
        TIMEMORY_BASIC_MARKER(auto_tuple_t, "[output]");
        std::cout << "Event: " << evt << std::endl;
        std::cout << _clock << std::endl;
        printf("Max error: %8.4e\n", maxError);
        printf("Sum error: %8.4e\n", sumError);
        printf("Effective Bandwidth (GB/s): %f\n",
               N * 4 * 3 / nseconds / tim::units::gigabyte);
        printf("Kernel Runtime (sec): %16.12e\n", nseconds);
    }

    tim::cuda::device_sync();
    // tim::cuda::device_reset();
}

//======================================================================================//
namespace impl
{
template <typename T>
TIMEMORY_GLOBAL_FUNCTION void
KERNEL_A(T* begin, int n)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        if(i < n)
            *(begin + i) += 2.0f * n;
    }
}

//--------------------------------------------------------------------------------------//

template <typename T>
TIMEMORY_GLOBAL_FUNCTION void
KERNEL_B(T* begin, int n)
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        if(i < n / 2)
            *(begin + i) *= 2.0f;
        else if(i >= n / 2 && i < n)
            *(begin + i) += 3.0f;
    }
}
}  // namespace impl
//--------------------------------------------------------------------------------------//

template <typename T>
void
KERNEL_A(T* arg, int size, tim::cuda::stream_t stream = 0)
{
    impl::KERNEL_A<<<2, 64, 0, stream>>>(arg, size);
}

//--------------------------------------------------------------------------------------//
template <typename T>
void
KERNEL_B(T* arg, int size, tim::cuda::stream_t stream = 0)
{
    impl::KERNEL_B<<<64, 2, 0, stream>>>(arg, size / 2);
}

//======================================================================================//
#if defined(TIMEMORY_USE_CUPTI)

static auto max_size = tim::get_env("MAX_SIZE", 64);
static auto num_data = tim::get_env("NUM_SIZE", 100);
static auto num_iter = tim::get_env("NUM_ITER", 10);

void
test_7_cupti_available()
{
    print_info(__FUNCTION__);

    hipDevice_t device;
    TIMEMORY_CUDA_DRIVER_API_CALL(hipInit(0));
    TIMEMORY_CUDA_DRIVER_API_CALL(hipDeviceGet(&device, 0));

    auto reduce_size = [](std::vector<std::string>& arr) {
        std::sort(arr.begin(), arr.end());
        if(arr.size() > (size_t) max_size)
            arr.resize(max_size);
    };

    auto event_names  = tim::cupti::available_events(device);
    auto metric_names = tim::cupti::available_metrics(device);
    reduce_size(event_names);
    reduce_size(metric_names);

    using size_type = decltype(event_names.size());
    size_type wevt  = 30;
    size_type wmet  = 30;
    for(const auto& itr : event_names)
        wevt = std::max(itr.size(), wevt);
    for(const auto& itr : metric_names)
        wmet = std::max(itr.size(), wmet);

    std::cout << "Event names: \n\t"
              << array_to_string(event_names, ", ", wevt, 180 / wevt) << std::endl;
    std::cout << "Metric names: \n\t"
              << array_to_string(metric_names, ", ", wmet, 180 / wmet) << std::endl;

    std::vector<float> cpu_data(num_data, 0);
    float*             data;
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMalloc(&data, num_data * sizeof(float)));
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMemcpy(
        data, cpu_data.data(), num_data * sizeof(float), hipMemcpyHostToDevice));

    tim::cupti::profiler profiler(event_names, metric_names);
    const int            passes = profiler.passes();
    printf("Passes: %d\n", passes);

    profiler.start();
    for(int i = 0; i < num_iter; ++i)
    {
        printf("\n[%s]> iteration %i...\n", __FUNCTION__, i);
        KERNEL_A(data, num_data);
        KERNEL_B(data, num_data);
    }
    tim::cuda::device_sync();
    profiler.stop();

    printf("Event Trace\n");
    profiler.print_event_values(std::cout);
    printf("Metric Trace\n");
    profiler.print_metric_values(std::cout);

    auto names = profiler.get_kernel_names();
    std::cout << "Kernel names: \n\t" << array_to_string(names, "\n\t", 16, names.size())
              << std::endl;

    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMemcpy(
        cpu_data.data(), data, num_data * sizeof(float), hipMemcpyDeviceToHost));
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipFree(data));

    printf("\n");
    std::cout << "Data values: \n\t" << array_to_string(cpu_data, ", ", 8, 10)
              << std::endl;
    printf("\n");
}

//======================================================================================//

void
test_8_cupti_subset()
{
    print_info(__FUNCTION__);
    // tim::cuda::device_reset();

    TIMEMORY_CUDA_DRIVER_API_CALL(hipInit(0));
    std::vector<std::string> event_names{ "active_warps",   "active_cycles",
                                          "global_load",    "global_store",
                                          "gld_inst_32bit", "gst_inst_32bit" };
    std::vector<std::string> metric_names{
        "inst_per_warp",     "branch_efficiency",  "warp_execution_efficiency",
        "flop_count_sp",     "flop_count_sp_add",  "flop_count_sp_fma",
        "flop_count_sp_mul", "flop_sp_efficiency", "gld_efficiency",
        "gst_efficiency"
    };

    std::vector<float> cpu_data(num_data, 0);
    float*             data;
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMalloc(&data, num_data * sizeof(float)));
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMemcpy(
        data, cpu_data.data(), num_data * sizeof(float), hipMemcpyHostToDevice));

    tim::cupti::profiler profiler(event_names, metric_names);
    // Get #passes required to compute all metrics and events
    const int passes = profiler.passes();
    printf("Passes: %d\n", passes);

    profiler.start();
    for(int i = 0; i < num_iter; ++i)
    {
        printf("\n[%s]> iteration %i...\n", __FUNCTION__, i);
        KERNEL_A(data, num_data);
        KERNEL_B(data, num_data);
    }
    tim::cuda::device_sync();
    profiler.stop();

    printf("Event Trace\n");
    profiler.print_event_values(std::cout);
    printf("Metric Trace\n");
    profiler.print_metric_values(std::cout);

    auto names = profiler.get_kernel_names();
    std::cout << "Kernel names: \n\t" << array_to_string(names, "\n\t", 16, names.size())
              << std::endl;

    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMemcpy(
        cpu_data.data(), data, num_data * sizeof(float), hipMemcpyDeviceToHost));
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipFree(data));

    printf("\n");
    std::cout << "Data values: \n\t" << array_to_string(cpu_data, ", ", 8, 10)
              << std::endl;
    printf("\n");
    // tim::cuda::device_reset();
}

//======================================================================================//

void
test_9_cupti_counters()
{
    print_info(__FUNCTION__);
    // tim::cuda::device_reset();

    hipDevice_t device;
    TIMEMORY_CUDA_DRIVER_API_CALL(hipInit(0));
    TIMEMORY_CUDA_DRIVER_API_CALL(hipDeviceGet(&device, 0));

    auto event_names  = tim::cupti::available_events(device);
    auto metric_names = tim::cupti::available_metrics(device);
    std::sort(event_names.begin(), event_names.end());
    std::sort(metric_names.begin(), metric_names.end());

    using size_type = decltype(event_names.size());
    size_type wevt  = 10;
    size_type wmet  = 10;
    for(const auto& itr : event_names)
        wevt = std::max(itr.size(), wevt);
    for(const auto& itr : metric_names)
        wmet = std::max(itr.size(), wmet);

    std::cout << "Event names: \n\t"
              << array_to_string(event_names, ", ", wevt, 200 / wevt) << std::endl;
    std::cout << "Metric names: \n\t"
              << array_to_string(metric_names, ", ", wmet, 200 / wmet) << std::endl;

    cupti_counters::get_device_initializer() = []() { return 0; };
    cupti_counters::get_event_initializer()  = []() {
        return std::vector<std::string>({ "active_warps", "active_cycles", "global_load",
                                          "global_store", "gld_inst_32bit",
                                          "gst_inst_32bit" });
    };
    cupti_counters::get_metric_initializer() = []() {
        return std::vector<std::string>({ "inst_per_warp", "branch_efficiency",
                                          "warp_execution_efficiency", "flop_count_sp",
                                          "flop_count_sp_add", "flop_count_sp_fma",
                                          "flop_count_sp_mul", "flop_sp_efficiency",
                                          "gld_efficiency", "gst_efficiency" });
    };

    using Tp        = double;
    using counter_t = tim::ert::counter<tim::device::gpu, Tp>;

    auto store_func = [] TIMEMORY_LAMBDA(Tp & a, const Tp& b) { a = b; };
    auto add_func   = [] TIMEMORY_LAMBDA(Tp & a, const Tp& b, const Tp& c) { a = b + c; };
    auto fma_func   = [] TIMEMORY_LAMBDA(Tp & a, const Tp& b, const Tp& c) {
        a = a * b + c;
    };

    tim::ert::exec_params       params(16, 64 * 64);
    std::shared_ptr<ert_data_t> exec_data(new ert_data_t);
    auto                        _counter = new counter_t(params, exec_data, 64);

    std::vector<float> cpu_data(num_data, 0);
    float*             data;
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMalloc(&data, num_data * sizeof(float)));
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMemcpy(
        data, cpu_data.data(), num_data * sizeof(float), hipMemcpyHostToDevice));

    cupti_counters profiler;
    profiler.start();
    for(int i = 0; i < num_iter; ++i)
    {
        printf("\n[%s]> iteration %i...\n", __FUNCTION__, i);
        tim::ert::ops_main<1>(*_counter, add_func, store_func);
        tim::ert::ops_main<2, 4, 8>(*_counter, fma_func, store_func);
        std::cout << *_counter << std::endl;
        KERNEL_A(data, num_data);
        KERNEL_B(data, num_data);
    }
    tim::cuda::device_sync();
    profiler.stop();
    std::cout << __FUNCTION__ << " : " << profiler << std::endl;

    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMemcpy(
        cpu_data.data(), data, num_data * sizeof(float), hipMemcpyDeviceToHost));
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipFree(data));

    printf("\n");
    // tim::cuda::device_reset();
}

//======================================================================================//

void
test_10_cupti_metric()
{
    print_info(__FUNCTION__);

    constexpr int      num_data = 100;
    std::vector<float> cpu_data(num_data, 0);
    float*             data;
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMalloc(&data, num_data * sizeof(float)));
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMemcpy(
        data, cpu_data.data(), num_data * sizeof(float), hipMemcpyHostToDevice));

    tim::cupti::profiler prof({ "gst_inst_32bit" }, { "flop_count_sp" });

    prof.start();
    for(int i = 0; i < 10; ++i)
    {
        KERNEL_A(data, num_data);
        KERNEL_B(data, num_data);
    }
    _LOG("calling sync...\n");
    tim::cuda::device_sync();
    prof.stop();

    TIMEMORY_CUDA_RUNTIME_API_CALL(hipMemcpy(
        cpu_data.data(), data, num_data * sizeof(float), hipMemcpyDeviceToHost));
    TIMEMORY_CUDA_RUNTIME_API_CALL(hipFree(data));

    printf("\n");
    std::cout << "Data values: \n\t" << array_to_string(cpu_data, ", ", 8, 10)
              << std::endl;
    printf("\n");
}

//======================================================================================//

#else  // defined(TIMEMORY_USE_CUPTI)

//======================================================================================//

void
test_7_cupti_available()
{
    print_info(__FUNCTION__);
    printf("CUPTI is not available...\n");
}

//======================================================================================//

void
test_8_cupti_subset()
{
    print_info(__FUNCTION__);
    printf("CUPTI is not available...\n");
}

//======================================================================================//

void
test_9_cupti_counters()
{
    print_info(__FUNCTION__);
    printf("CUPTI is not available...\n");
}

//======================================================================================//

void
test_10_cupti_metric()
{
    print_info(__FUNCTION__);
    printf("CUPTI is not available...\n");
}

//======================================================================================//

#endif  // defined(TIMEMORY_USE_CUPTI)
